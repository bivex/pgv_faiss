
#include <hip/hip_runtime.h>
#ifdef WITH_GPU

#include "faiss_wrapper.h"
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuIndexIVFFlat.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <cuda_runtime.h>
#include <iostream>

class FAISSWrapper::GPUImpl {
public:
    std::unique_ptr<faiss::gpu::StandardGpuResources> gpu_resources;
    int gpu_device;
    
    GPUImpl(int device) : gpu_device(device) {
        setup_gpu_resources();
    }
    
    void setup_gpu_resources() {
        // TODO: Add GPU capability checking (compute capability, memory size)
        // TODO: Implement GPU memory optimization strategies
        // TODO: Add support for multiple GPU devices and load balancing
        // TODO: Implement GPU memory monitoring and adaptive allocation
        
        if (faiss::gpu::getNumDevices() <= gpu_device) {
            std::cerr << "GPU device " << gpu_device << " not available" << std::endl;
            return;
        }
        
        cudaError_t err = cudaSetDevice(gpu_device);
        if (err != cudaSuccess) {
            std::cerr << "Failed to set CUDA device " << gpu_device 
                      << ": " << cudaGetErrorString(err) << std::endl;
            return;
        }
        
        try {
            gpu_resources = std::make_unique<faiss::gpu::StandardGpuResources>();
            
            size_t free_mem, total_mem;
            cudaMemGetInfo(&free_mem, &total_mem);
            
            // TODO: Make memory allocation strategy configurable
            // TODO: Add memory pool management for better performance
            size_t temp_mem = std::min(free_mem / 4, size_t(1536 * 1024 * 1024));
            gpu_resources->setTempMemory(temp_mem);
            
            std::cout << "GPU " << gpu_device << " initialized with " 
                      << temp_mem / (1024*1024) << " MB temp memory" << std::endl;
                      
        } catch (const std::exception& e) {
            std::cerr << "Failed to initialize GPU resources: " << e.what() << std::endl;
            gpu_resources.reset();
        }
    }
    
    faiss::Index* create_gpu_index(const std::string& index_type, int dimension) {
        if (!gpu_resources) {
            return nullptr;
        }
        
        // TODO: Add automatic GPU index selection based on data size and GPU memory
        // TODO: Implement fallback to CPU when GPU memory is insufficient
        // TODO: Add support for more GPU index types (GpuIndexIVFPQ, etc.)
        // TODO: Implement index parameter optimization based on GPU architecture
        
        try {
            if (index_type == "Flat") {
                return new faiss::gpu::GpuIndexFlat(
                    gpu_resources.get(), dimension, faiss::METRIC_L2);
            }
            else if (index_type == "IVFFlat") {
                // TODO: Make ncentroids adaptive based on dataset size and GPU memory
                int ncentroids = std::min(4 * (int)sqrt(100000), 65536);
                return new faiss::gpu::GpuIndexIVFFlat(
                    gpu_resources.get(), dimension, ncentroids, faiss::METRIC_L2);
            }
        } catch (const std::exception& e) {
            std::cerr << "Failed to create GPU index: " << e.what() << std::endl;
        }
        
        return nullptr;
    }
    
    void print_gpu_info() {
        int device_count;
        cudaGetDeviceCount(&device_count);
        
        std::cout << "Found " << device_count << " CUDA devices:" << std::endl;
        
        for (int i = 0; i < device_count; ++i) {
            cudaDeviceProp prop;
            cudaGetDeviceProperties(&prop, i);
            
            std::cout << "  Device " << i << ": " << prop.name 
                      << " (" << prop.major << "." << prop.minor << ")" << std::endl;
            std::cout << "    Memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
            std::cout << "    Multiprocessors: " << prop.multiProcessorCount << std::endl;
        }
    }
};

void FAISSWrapper::setup_gpu_resources() {
    if (use_gpu_) {
        gpu_impl_ = std::make_unique<GPUImpl>(gpu_device_);
    }
}

faiss::Index* FAISSWrapper::create_gpu_index(const std::string& index_type, int dimension) {
    if (use_gpu_ && gpu_impl_) {
        return gpu_impl_->create_gpu_index(index_type, dimension);
    }
    return nullptr;
}

void FAISSWrapper::print_gpu_info() {
    if (gpu_impl_) {
        gpu_impl_->print_gpu_info();
    }
}

__global__ void warm_up_gpu() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    volatile float dummy = sinf(float(idx));
}

void FAISSWrapper::warm_up_gpu() {
    if (use_gpu_) {
        cudaSetDevice(gpu_device_);
        warm_up_gpu<<<256, 256>>>();
        cudaDeviceSynchronize();
    }
}

#endif